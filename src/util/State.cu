#include "State.cuh"
#include <algorithm>

State::State(uint64_t p, int observations, double alpha, int maxCondSize)
    : p(p), observations(observations), alpha(alpha), maxCondSize(maxCondSize) {
  cor = reinterpret_cast<double *>(malloc(p * p * sizeof(double)));
  adj = reinterpret_cast<int *>(malloc(p * p * sizeof(int)));
  std::fill_n(adj, p * p, 1);
  for (int i = 0; i < p; ++i) {
    adj[i * p + i] = 0;
  }
  pMax = reinterpret_cast<double *>(malloc(p * p * sizeof(double)));
  std::fill_n(pMax, p * p, 0.0);
  sepSets = reinterpret_cast<int *>(malloc(maxCondSize * p * p * sizeof(int)));
  std::fill_n(sepSets, p * p * maxCondSize, -1);
  hipHostRegister(pMax, (uint64_t)sizeof(double) * p * p, 0);
  hipHostRegister(cor, (uint64_t)sizeof(double) * p * p, 0);
  hipHostRegister(adj, (uint64_t)sizeof(int) * p * p, 0);
  hipHostRegister(sepSets, (uint64_t)sizeof(int) * p * p * maxCondSize, 0);
}

State::State(const State &state)
    : p(state.p), observations(state.observations), alpha(state.alpha),
      maxCondSize(state.maxCondSize) {
  adj = new int[p * p];
  memcpy(adj, state.adj, p * p * sizeof(int));
  pMax = new double[p * p];
  memcpy(pMax, state.pMax, p * p * sizeof(double));
  cor = new double[p * p];
  memcpy(cor, state.cor, p * p * sizeof(double));
  sepSets = new int[p * p * maxCondSize];
  memcpy(sepSets, state.sepSets, p * p * maxCondSize * sizeof(int));
  hipHostRegister(pMax, (uint64_t)sizeof(double) * p * p, 0);
  hipHostRegister(cor, (uint64_t)sizeof(double) * p * p, 0);
  hipHostRegister(adj, (uint64_t)sizeof(int) * p * p, 0);
  hipHostRegister(sepSets, (uint64_t)sizeof(int) * p * p * maxCondSize, 0);
}

State::~State() {
  hipHostUnregister(pMax);
  hipHostUnregister(cor);
  hipHostUnregister(adj);
  hipHostUnregister(sepSets);
  delete[] pMax;
  delete[] sepSets;
  delete[] cor;
  delete[] adj;
}

GPUState::GPUState(uint64_t p, int observations, double alpha, int maxCondSize)
    : p(p), observations(observations), alpha(alpha), maxCondSize(maxCondSize) {
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&adj),
                             (uint64_t)sizeof(int) * p * p));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&cor),
                             (uint64_t)sizeof(double) * p * p));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&pMax),
                             (uint64_t)sizeof(double) * p * p));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&sepSets),
                             (uint64_t)sizeof(int) * p * p * maxCondSize));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&lock),
                             (uint64_t)sizeof(int) * p * p));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&rowMapping),
                             (uint64_t)sizeof(int) * p));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&colMapping),
                             (uint64_t)sizeof(int) * p));
  checkCudaErrors(hipMemset(lock, 0, (uint64_t)sizeof(int) * p * p));
}

void GPUState::destroy() {
  checkCudaErrors(hipFree(adj));
  checkCudaErrors(hipFree(cor));
  checkCudaErrors(hipFree(pMax));
  checkCudaErrors(hipFree(sepSets));
  checkCudaErrors(hipFree(lock));
  checkCudaErrors(hipFree(rowMapping));
  checkCudaErrors(hipFree(colMapping));
}

MMGPUState::MMGPUState(uint64_t p, int observations, double alpha,
                       int maxCondSize)
    : p(p), observations(observations), alpha(alpha), maxCondSize(maxCondSize) {
  checkCudaErrors(hipMallocManaged(&adj, (uint64_t)sizeof(int) * p * p));
  checkCudaErrors(hipMallocManaged(&cor, (uint64_t)sizeof(double) * p * p));
  checkCudaErrors(hipMallocManaged(&pMax, (uint64_t)sizeof(double) * p * p));
  checkCudaErrors(
      hipMallocManaged(&sepSets, (uint64_t)sizeof(int) * p * p * maxCondSize));
  checkCudaErrors(
      hipMallocManaged(&adj_compact, (uint64_t)sizeof(int) * p * p));
  checkCudaErrors(hipMallocManaged(&max_adj, (uint64_t)sizeof(int)));
  checkCudaErrors(hipMallocManaged(&lock, (uint64_t)sizeof(int) * p * p));
  std::fill_n(adj, p * p, 1);
  std::fill_n(adj_compact, p * p, 1);
  for (int i = 0; i < p; ++i) {
    adj[i * p + i] = 0;
    adj_compact[i * p + i] = 0;
  }
  std::fill_n(pMax, p * p, 0.0);
  std::fill_n(sepSets, p * p * maxCondSize, -1);

  memset(lock, 0, (uint64_t)sizeof(int) * p * p);
  max_adj[0] = p;
}

void MMGPUState::destroy() {
  checkCudaErrors(hipFree(adj));
  checkCudaErrors(hipFree(cor));
  checkCudaErrors(hipFree(pMax));
  checkCudaErrors(hipFree(sepSets));
  checkCudaErrors(hipFree(max_adj));
  checkCudaErrors(hipFree(adj_compact));
  checkCudaErrors(hipFree(lock));
}
