#include "hip/hip_runtime.h"
#include "state.cuh"
#include <algorithm>
#include <iostream>

MMState::MMState(uint64_t p, int observations, double alpha, int maxLevel, int mainDeviceId)
    : p(p), observations(observations), alpha(alpha), maxLevel(maxLevel)
{
  checkCudaErrors(hipSetDevice(mainDeviceId));
  maxCondSize = std::max(maxLevel, 1);
  checkCudaErrors(hipMallocManaged(&adj, (uint64_t)sizeof(int) * p * p));
  checkCudaErrors(hipMallocManaged(&cor, (uint64_t)sizeof(double) * p * p));
  checkCudaErrors(hipMallocManaged(&pMax, (uint64_t)sizeof(double) * p * p));
  checkCudaErrors(hipMallocManaged(&node_status, (uint64_t)sizeof(bool) * p * p));
  checkCudaErrors(
      hipMallocManaged(&sepSets, (uint64_t)sizeof(int) * p * p * maxCondSize));
  checkCudaErrors(
      hipMallocManaged(&adj_compact, (uint64_t)sizeof(int) * p * p));
  checkCudaErrors(hipMallocManaged(&max_adj, (uint64_t)sizeof(int)));
  checkCudaErrors(hipMallocManaged(&lock, (uint64_t)sizeof(int) * p * p));
  std::fill_n(adj, p * p, 1);
  std::fill_n(adj_compact, p * p, 1);
  std::fill_n(node_status, p * p, false);
  for (int i = 0; i < p; ++i)
  {
    adj[i * p + i] = 0;
    adj_compact[i * p + i] = 0;
  }
  std::fill_n(pMax, p * p, 0.0);
  std::fill_n(sepSets, p * p * maxCondSize, -1);
  memset(lock, 0, (uint64_t)sizeof(int) * p * p);
  max_adj[0] = (int)p;
  gpu_done = false;
}

void MMState::adviceReadonlyCor(std::vector<int> gpuList)
{
  checkCudaErrors(hipMemAdvise(cor, (uint64_t)sizeof(double) * p * p, hipMemAdviseSetReadMostly, 0));
  for (auto deviceId : gpuList)
  {
    checkCudaErrors(hipMemPrefetchAsync(cor,
                                         (uint64_t)sizeof(double) * p * p, deviceId));
  }
}

void MMState::prefetchRows(int startRow, int rowCount, int deviceId)
{
  checkCudaErrors(hipMemPrefetchAsync(adj + startRow * p,
                                       (uint64_t)sizeof(int) * rowCount,
                                       deviceId));
  checkCudaErrors(hipMemPrefetchAsync(adj_compact + startRow * p,
                                       (uint64_t)sizeof(int) * rowCount,
                                       deviceId));
  checkCudaErrors(hipMemPrefetchAsync(pMax + startRow * p,
                                       (uint64_t)sizeof(double) * rowCount,
                                       deviceId));
  checkCudaErrors(hipMemPrefetchAsync(sepSets + startRow * p * maxCondSize,
                                       (uint64_t)sizeof(int) * rowCount * maxCondSize,
                                       deviceId));
}

void MMState::memAdvise(std::vector<int> gpuList)
{
  checkCudaErrors(hipMemAdvise(adj_compact,
                                (uint64_t)sizeof(int) * p * p,
                                hipMemAdviseSetReadMostly, 0));

  int numberOfGPUs = gpuList.size();
  for (int i = 0; i < gpuList.size(); i++)
  {
    int deviceId = gpuList[i];

    checkCudaErrors(hipMemAdvise(lock + ((p * p / numberOfGPUs) * i),
                                  (uint64_t)sizeof(int) * p * p / numberOfGPUs,
                                  hipMemAdviseSetPreferredLocation, deviceId));

    checkCudaErrors(hipMemAdvise(lock,
                                  (uint64_t)sizeof(int) * p * p,
                                  hipMemAdviseSetAccessedBy, deviceId));
  }
}

void MMState::destroy()
{
  checkCudaErrors(hipFree(adj));
  checkCudaErrors(hipFree(cor));
  checkCudaErrors(hipFree(pMax));
  checkCudaErrors(hipFree(sepSets));
  checkCudaErrors(hipFree(adj_compact));
  checkCudaErrors(hipFree(lock));
  checkCudaErrors(hipFree(node_status));
}