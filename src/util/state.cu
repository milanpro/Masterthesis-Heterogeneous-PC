#include "hip/hip_runtime.h"
#include "state.cuh"
#include <algorithm>

MMState::MMState(uint64_t p, int observations, double alpha, int maxLevel)
    : p(p), observations(observations), alpha(alpha), maxLevel(maxLevel) {
  maxCondSize = std::max(maxLevel, 1);
  checkCudaErrors(hipMallocManaged(&adj, (uint64_t)sizeof(int) * p * p));
  checkCudaErrors(hipMallocManaged(&cor, (uint64_t)sizeof(double) * p * p));
  checkCudaErrors(hipMallocManaged(&pMax, (uint64_t)sizeof(double) * p * p));
  checkCudaErrors(
  hipMallocManaged(&sepSets, (uint64_t)sizeof(int) * p * p * maxCondSize));
  checkCudaErrors(
  hipMallocManaged(&adj_compact, (uint64_t)sizeof(int) * p * p));
  checkCudaErrors(hipMallocManaged(&max_adj, (uint64_t)sizeof(int)));
  checkCudaErrors(hipMallocManaged(&lock, (uint64_t)sizeof(int) * p * p));
  std::fill_n(adj, p * p, 1);
  std::fill_n(adj_compact, p * p, 1);
  for (int i = 0; i < p; ++i) {
    adj[i * p + i] = 0;
    adj_compact[i * p + i] = 0;
  }
  std::fill_n(pMax, p * p, 0.0);
  std::fill_n(sepSets, p * p * maxCondSize, -1);
  memset(lock, 0, (uint64_t)sizeof(int) * p * p);
  max_adj[0] = (int)p;
}

void MMState::adviceReadonlyCor(int numberOfGPUs) {
  checkCudaErrors(hipMemAdvise(cor, (uint64_t)sizeof(double) * p * p, hipMemAdviseSetReadMostly, 0));
  for (int deviceId = 0; deviceId < numberOfGPUs; deviceId++) {
    checkCudaErrors(hipMemPrefetchAsync(cor,
      (uint64_t)sizeof(double) * p * p, deviceId));
  }
}

void MMState::prefetchRows(int startRow, int rowCount, int deviceId) {
      checkCudaErrors(hipMemPrefetchAsync(adj + startRow * p,
        (uint64_t)sizeof(int) * rowCount,
        deviceId, 0));
      checkCudaErrors(hipMemPrefetchAsync(adj_compact + startRow * p,
        (uint64_t)sizeof(int) * rowCount,
        deviceId, 0));
      checkCudaErrors(hipMemPrefetchAsync(pMax + startRow * p,
        (uint64_t)sizeof(double) * rowCount,
        deviceId, 0));
      checkCudaErrors(hipMemPrefetchAsync(sepSets + startRow * p * maxCondSize,
        (uint64_t)sizeof(int) * rowCount * maxCondSize,
        deviceId, 0));
    }

void MMState::memAdvise(int numberOfGPUs) {
  checkCudaErrors(hipMemAdvise(adj_compact,
    (uint64_t)sizeof(int) * p * p,
    hipMemAdviseSetReadMostly, 0));

  for (int deviceId = 0; deviceId < numberOfGPUs; deviceId++) {
//     checkCudaErrors(hipMemAdvise(adj + p * p / numberOfGPUs * deviceId,
//       (uint64_t)sizeof(int) * p * p / numberOfGPUs,
//       hipMemAdviseSetPreferredLocation, deviceId));

// checkCudaErrors(hipMemAdvise(pMax + p * p / numberOfGPUs * deviceId,
//       (uint64_t)sizeof(double) * p * p / numberOfGPUs,
//       hipMemAdviseSetPreferredLocation, deviceId));

// checkCudaErrors(hipMemAdvise(sepSets + p * p * maxCondSize / numberOfGPUs * deviceId,
//       (uint64_t)sizeof(int) * p * p * maxCondSize / numberOfGPUs,
//       hipMemAdviseSetPreferredLocation, deviceId));

checkCudaErrors(hipMemAdvise(lock + p * p / numberOfGPUs * deviceId,
      (uint64_t)sizeof(int) * p * p / numberOfGPUs,
      hipMemAdviseSetPreferredLocation, deviceId));

// setting accessed by
// checkCudaErrors(hipMemAdvise(adj,
//       (uint64_t)sizeof(int) * p * p,
//       hipMemAdviseSetAccessedBy, deviceId));
// checkCudaErrors(hipMemAdvise(adj_compact,
//       (uint64_t)sizeof(int) * p * p,
//       hipMemAdviseSetAccessedBy, deviceId));
// checkCudaErrors(hipMemAdvise(pMax,
//       (uint64_t)sizeof(double) * p * p,
//       hipMemAdviseSetAccessedBy, deviceId));
// checkCudaErrors(hipMemAdvise(sepSets,
//       (uint64_t)sizeof(int) * p * p * maxCondSize,
//       hipMemAdviseSetAccessedBy, deviceId));
checkCudaErrors(hipMemAdvise(lock,
      (uint64_t)sizeof(int) * p * p,
      hipMemAdviseSetAccessedBy, deviceId));

  }
}

void MMState::destroy() {
  checkCudaErrors(hipFree(adj));
  checkCudaErrors(hipFree(cor));
  checkCudaErrors(hipFree(pMax));
  checkCudaErrors(hipFree(sepSets));
  checkCudaErrors(hipFree(adj_compact));
  checkCudaErrors(hipFree(lock));
}