#include "hip/hip_runtime.h"
#include "gpuWorkstealingTests.cuh"
#include "../independence/gpuInd.cuh"

__global__ void testRowWorkstealingL1(MMState state, int *rows, int start_row, int max_row_count)
{
  if (start_row + blockIdx.x >= max_row_count)
  {
    return;
  }

  size_t row_node = rows[start_row + blockIdx.x];
  size_t row_neighbours = state.adj_compact[row_node * state.p + state.p - 1];

  size_t col_node = state.adj_compact[row_node * state.p + blockIdx.y];

  if (row_node == col_node)
  {
    return;
  }

  __shared__ bool active;
  if (threadIdx.x == 0)
  {
#if WITH_CUDA_ATOMICS
    bool expected = false;
    active = state.node_status[row_node * state.p + col_node].compare_exchange_strong(expected, true);
#else
    active = !state.node_status[row_node * state.p + col_node];
    if (active)
    {
      state.node_status[row_node * state.p + col_node] = true;
    }
#endif
  }
  __syncthreads();
  if (!active)
  {
    return;
  }

  extern __shared__ double pVals[];
  if (row_neighbours > blockIdx.y && row_neighbours >= 1)
  {
    size_t subIndex = 0;
    for (size_t offset = threadIdx.x; offset < row_neighbours; offset += blockDim.x)
    {
      if (offset == blockIdx.y)
      {
        pVals[threadIdx.x] = -1;
      }
      else
      {
        subIndex = state.adj_compact[row_node * state.p + offset];
        pVals[threadIdx.x] = GPU::pValL1(
            state.cor[row_node * state.p + col_node],
            state.cor[row_node * state.p + subIndex],
            state.cor[col_node * state.p + subIndex], state.observations);
      }

      __syncthreads();
      if (threadIdx.x == 0)
      {
        for (size_t i = 0; i < blockDim.x && i < row_neighbours; ++i)
        {
          double pVal = pVals[i];
          if (offset + i < state.p && pVal >= state.alpha)
          {
            state.node_status[col_node * state.p + row_node] = true;
            if (row_node < col_node)
            {
              if (atomicCAS_system(&state.lock[(state.p * row_node) + col_node], 0, 1) == 0)
              {
                state.adj[row_node * state.p + col_node] = 0;
                state.adj[col_node * state.p + row_node] = 0;
                state.sepSets[row_node * state.p * state.maxCondSize +
                              col_node * state.maxCondSize] =
                    state.adj_compact[row_node * state.p + offset + i];
                state.pMax[state.p * row_node + col_node] = pVal;
              }
            }
            else
            {
              if (atomicCAS_system(&state.lock[(state.p * col_node) + row_node], 0, 1) == 0)
              {
                state.adj[row_node * state.p + col_node] = 0;
                state.adj[col_node * state.p + row_node] = 0;
                state.pMax[state.p * col_node + row_node] = pVal;
                state.sepSets[col_node * state.p * state.maxCondSize +
                              row_node * state.maxCondSize] =
                    state.adj_compact[col_node * state.p + offset + i];
              }
            }
            break;
          }
        }
      }
      __syncthreads();
      if (state.adj[row_node * state.p + col_node] == 0)
        break;
    }
  }
}

template <int lvlSize, int kLvlSizeSmall>
__global__ void testRowWorkstealingLN(MMState state, int *rows, int start_row, int max_row_count)
{
  if (start_row + blockIdx.x >= max_row_count)
  {
    return;
  }

  size_t row_node = rows[start_row + blockIdx.x];
  size_t row_count = state.adj_compact[row_node * state.p + state.p - 1];
  if (row_count > blockIdx.y && // col_node available
      row_count >= kLvlSizeSmall)
  {
    size_t col_node = state.adj_compact[row_node * state.p + blockIdx.y]; // get actual id

    if (row_node == col_node)
    {
      return;
    }

    __shared__ bool active;
    if (threadIdx.x == 0)
    {
#if WITH_CUDA_ATOMICS
      bool expected = false;
      active = state.node_status[row_node * state.p + col_node].compare_exchange_strong(expected, true);
#else
      active = !state.node_status[row_node * state.p + col_node];
      if (active)
      {
        state.node_status[row_node * state.p + col_node] = true;
      }
#endif
    }
    __syncthreads();
    if (!active)
    {
      return;
    }

    double Submat[lvlSize][lvlSize];
    double SubmatPInv[lvlSize][lvlSize];
    int sepset_nodes[kLvlSizeSmall];
    // pseudo-inverse parameter
    double v[lvlSize][lvlSize];
    double w[lvlSize], rv1[lvlSize];
    double res1[lvlSize][lvlSize];
    // Determine sepsets to work on

    int row_neighbours = row_count - 1; // get number of neighbours && exclude col_node
    size_t row_test_count = binomialCoeff(row_neighbours, kLvlSizeSmall);
    for (size_t test_index = threadIdx.x; test_index < row_test_count;
         test_index += blockDim.x)
    {
      ithCombination(sepset_nodes, test_index, kLvlSizeSmall,
                     row_neighbours);
      for (int ind = 0; ind < kLvlSizeSmall; ++ind)
      {
        if (sepset_nodes[ind] - 1 >= blockIdx.y)
        {
          sepset_nodes[ind] =
              state.adj_compact[row_node * state.p + sepset_nodes[ind]];
        }
        else
        {
          sepset_nodes[ind] =
              state.adj_compact[row_node * state.p + sepset_nodes[ind] - 1];
        }
      }
      for (int i = 0; i < lvlSize; ++i)
      {
        // set diagonal
        Submat[i][i] = 1;
      }
      Submat[0][1] = Submat[1][0] = state.cor[row_node * state.p + col_node];
      for (int j = 2; j < lvlSize; ++j)
      {
        // set correlations of X
        Submat[0][j] = Submat[j][0] =
            state.cor[row_node * state.p + sepset_nodes[j - 2]];
        // set correlations of Y
        Submat[1][j] = Submat[j][1] =
            state.cor[col_node * state.p + sepset_nodes[j - 2]];
      }
      for (int i = 2; i < lvlSize; ++i)
      {
        for (int j = i + 1; j < lvlSize; ++j)
        {
          Submat[i][j] = Submat[j][i] =
              state.cor[sepset_nodes[i - 2] * state.p + sepset_nodes[j - 2]];
        }
      }
      if (state.adj[state.p * row_node + col_node] == 0)
      {
        break;
      }
      pseudoinverse<lvlSize>(Submat, SubmatPInv, v, rv1, w, res1);
      double r = -SubmatPInv[0][1] / sqrt(SubmatPInv[0][0] * SubmatPInv[1][1]);
      double pVal = GPU::calcPValue(r, state.observations);

      if (pVal >= state.alpha)
      {
        state.node_status[col_node * state.p + row_node] = true;
        if (row_node < col_node)
        {
          if (atomicCAS(&state.lock[(state.p * row_node) + col_node], 0, 1) == 0)
          {
            state.adj[state.p * row_node + col_node] = 0;
            state.adj[state.p * col_node + row_node] = 0;
            state.pMax[state.p * row_node + col_node] = pVal;
            for (int j = 0; j < kLvlSizeSmall; ++j)
            {
              state.sepSets[row_node * state.p * state.maxCondSize +
                            col_node * state.maxCondSize + j] = sepset_nodes[j];
            }
          }
        }
        else
        {
          if (atomicCAS(&state.lock[(state.p * col_node) + row_node], 0, 1) == 0)
          {
            state.adj[state.p * row_node + col_node] = 0;
            state.adj[state.p * col_node + row_node] = 0;
            state.pMax[state.p * col_node + row_node] = pVal;
            for (int j = 0; j < kLvlSizeSmall; ++j)
            {
              state.sepSets[col_node * state.p * state.maxCondSize +
                            row_node * state.maxCondSize + j] = sepset_nodes[j];
            }
          }
        }
      }
    }
  }
}

template __global__ void testRowWorkstealingLN<4, 2>(MMState state, int *rows, int start_row, int max_row_count);

template __global__ void testRowWorkstealingLN<5, 3>(MMState state, int *rows, int start_row, int max_row_count);
