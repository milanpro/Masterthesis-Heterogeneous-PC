#include "hip/hip_runtime.h"
#include "../util/cudaUtil.cuh"
#include "../util/matrixPrint.cuh"
#include "../util/constants.hpp"
#include "cpu.cuh"
#include "gpu.cuh"
#include "skeleton.cuh"
#include "mm_test.cuh"
#include <iostream>
#include <string>
#include <unordered_set>
#include <memory>

void calcSkeleton(MMGPUState *state, int gpusUsed, int maxMem,
                    int startLevel) {

  int devID = setCudaDevice();
  int maxEdgeCount = state->p * (state->p - 1L) / 2;
  if (VERBOSE)
    std::cout << "maxCondSize: " << state->maxCondSize
              << "  observations: " << state->observations
              << "  p: " << state->p << " gpusUsed: " << gpusUsed << std::endl;

  TestResult res, resCPU, resGPU;
  if (startLevel <= 0) {
    auto cpuQueue = std::unique_ptr<SplitTaskQueue>(new SplitTaskQueue());
    auto gpuQueue = std::unique_ptr<SplitTaskQueue>(new SplitTaskQueue());
    for (int row = 0; row < state->p; row++)
    {
      if (row % 2 == 0) {
        cpuQueue->enqueue(SplitTask{row});
      } else {
        gpuQueue->enqueue(SplitTask{row});
      }
    }
    
    resCPU = cpuIndTest(0, state, cpuQueue.get());
    resGPU = gpuIndTest(0, state, gpuQueue.get(), maxEdgeCount);
    if (VERBOSE) {
      std::cout << "Order 0 finished with " << resCPU.tests + resGPU.tests << " tests in "
                << max(resCPU.duration,resGPU.duration) << " µs." << std::endl;
      std::cout << "\t CPU time: " << resCPU.duration  << " µs GPU time: "
                << resGPU.duration << " µs." << std::endl;
    }
  }

  if (state->maxCondSize < 1) {
    return;
  }
  if (startLevel <= 1) {
    auto cpuQueue = std::unique_ptr<SplitTaskQueue>(new SplitTaskQueue());
    auto gpuQueue = std::unique_ptr<SplitTaskQueue>(new SplitTaskQueue());
    for (int row = 0; row < state->p; row++)
    {
      if (row % 2 == 0) {
        cpuQueue->enqueue(SplitTask{row});
      } else {
        gpuQueue->enqueue(SplitTask{row});
      }
    }
    
    resCPU = cpuIndTest(1, state, cpuQueue.get());
    resGPU = gpuIndTest(1, state, gpuQueue.get(), maxEdgeCount);
    if (VERBOSE) {
      std::cout << "Order 1 finished with " << resCPU.tests + resGPU.tests << " tests in "
                << max(resCPU.duration,resGPU.duration) << " µs." << std::endl;
      std::cout << "\t CPU time: " << resCPU.duration  << " µs GPU time: "
                << resGPU.duration << " µs." << std::endl;
    }
  }


  int lvl = (startLevel < 2) ? 2 : startLevel;
  while(lvl <= state->maxCondSize){
    res = MMtestLN(state, maxMem, gpusUsed, lvl);
    if (VERBOSE) {
      std::cout << "Order " << lvl << " finished with " << res.tests
                << " tests in " << res.duration << " microseconds."
                << std::endl;
    }
    ++lvl;
  }

  if (VERBOSE) {
    printMMSepsets(state);
  }
}

void printMMSepsets(MMGPUState *state) {
  int nrEdges = 0;
  for (int i = 0; i < state->p; i++) {
    for (int j = i + 1; j < state->p; j++) {
      if (!state->adj[i * state->p + j]) {
        std::string sepset_string = "";
        for (int k = 0; k < state->maxCondSize; k++) {
          int current_sepset_node =
              state->sepSets[(i * state->maxCondSize * state->p) +
                             (j * state->maxCondSize) + k];
          if (current_sepset_node == -2) {
            std::cout << "Separation from " << i << " to " << j << std::endl;
            break;
          } else if (current_sepset_node == -1) {
            break;
          } else {
            sepset_string.append(std::to_string(current_sepset_node));
            sepset_string.append(" ");
          }
        }
        if (sepset_string != "") {
          std::cout << "Separation from " << i << " to " << j << " via "
                    << sepset_string << std::endl;
        }
      } else {
        nrEdges++;
      }
    }
  }
  std::cout << "Total number of edges: " << nrEdges << std::endl;
}
